#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_edge_unweighted_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SigmoidCrossEntropyEdgeUnweightedLossCountGPU(const int nthreads,
          const Dtype* target, Dtype* counts, const int cha, const int dim) {
  CUDA_KERNEL_LOOP (i, nthreads) {
    int n = i / dim;
    int d = i % dim;
    bool pflag = false, nflag = false;
    for (int k = 0; k < cha; k++) {
      const int target_value = static_cast<int>(target[n*cha*dim + k*dim + d]);
      pflag = pflag || (target_value == 1);
      nflag = nflag || (target_value == 0);
    }
    if (pflag) {
      counts[n*cha*dim + 0*dim + d] = 1;
    }
    else if (nflag) {
      counts[n*cha*dim + 0*dim + d] = 0;
    }
    else {
      counts[n*cha*dim + 0*dim + d] = 255;
    }
  }
}

template <typename Dtype>
__global__ void SigmoidCrossEntropyEdgeUnweightedLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* target, Dtype* loss) {
  CUDA_KERNEL_LOOP (i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value == 1) {
      loss[i] = input_data[i] * (target[i] - (input_data[i] >= 0)) -
        log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));
    }
    else if (target_value == 0) {
      loss[i] = input_data[i] * (target[i] - (input_data[i] >= 0)) -
        log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));
    }
  }
}

template <typename Dtype>
__global__ void SigmoidCrossEntropyEdgeUnweightedLossNormlizeDiffGPU(const int nthreads,
    const Dtype* target, Dtype* diff) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value != 1 && target_value != 0) {
      diff[i] = 0;
    }
  }
}

template <typename Dtype>
void SigmoidCrossEntropyEdgeUnweightedLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int num = bottom[1]->num();
  const int cha = bottom[1]->shape(1);
  const int dim = bottom[1]->count(2);
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* count_data = bottom[1]->mutable_gpu_diff();

  SigmoidCrossEntropyEdgeUnweightedLossCountGPU<<<CAFFE_GET_BLOCKS(num*dim),
      CAFFE_CUDA_NUM_THREADS>>>(num*dim, target, count_data, cha, dim);

  Dtype loss = 0;
  for (int i = 0; i < num; i++) {
    SigmoidCrossEntropyEdgeUnweightedLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(dim),
        CAFFE_CUDA_NUM_THREADS>>>(dim, input_data + i*dim,
        count_data + i*cha*dim, loss_data + i*dim);
    Dtype temp_loss;
    caffe_gpu_asum(dim, loss_data + i*dim, &temp_loss);
    loss += temp_loss;
  }
  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
void SigmoidCrossEntropyEdgeUnweightedLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[1]->num();
    const int cha = bottom[1]->shape(1);
    const int dim = bottom[1]->count(2);
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* count_data = bottom[1]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);

    for (int i = 0; i < num; i++) {
      caffe_gpu_axpy(dim, Dtype(-1), count_data + i*cha*dim, bottom_diff + i*dim);
      SigmoidCrossEntropyEdgeUnweightedLossNormlizeDiffGPU<Dtype><<<CAFFE_GET_BLOCKS(dim),
          CAFFE_CUDA_NUM_THREADS>>>(dim, count_data + i*cha*dim,
          bottom_diff + i*dim);
    }

    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count, loss_weight / num, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyEdgeUnweightedLossLayer);

}  // namespace caffe
